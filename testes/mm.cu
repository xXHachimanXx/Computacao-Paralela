
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vecadd_cuda(double* a, double* b, double* c, int width) {
	  int i = blockIdx.x*blockDim.x+threadIdx.x;

	    if (i < width)
		    c[i] = a[i] + b[i];
}

int main()
{
	  int width = 10000000;
	  double *a = (double*) malloc (width * sizeof(double));
	  double *b = (double*) malloc (width * sizeof(double));
	  double *c = (double*) malloc (width * sizeof(double));

	  for(int i = 0; i < width; i++) {
	     a[i] = i;
	     b[i] = width-i;
	     c[i] = 0;
	  }

	  int size = width*sizeof(double);
	  double *d_a, *d_b, *d_c;
      hipMalloc((void **) &d_a, size);
	  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

      hipMalloc((void **) &d_b, size);
	  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

      hipMalloc((void **) &d_c, size);

	  int block_size = 1024;
      dim3 dimGrid((width-1)/block_size + 1, 1, 1);
	  dim3 dimBlock(block_size,1,1);

      vecadd_cuda<<<dimGrid,dimBlock>>>(d_a, d_b, d_c, width);

      hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	  for(int i = 0; i < width; i++)
	    printf("\n c[%d] = %f",i,c[i]);

		hipFree(d_a);
		hipFree(d_b);
		hipFree(d_c);

}
