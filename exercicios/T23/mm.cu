#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
/*
 * Tempo Sequencial:
 * real    1m9.236s
 * user    1m8.004s
 * sys     0m0.132s
 *
 * Tempo multicore:
 * real    0m17.290s
 * user    1m8.421s
 * sys     0m0.148s
 *
 * Tempo, warps_launched e warp_execution_efficiency GPU OpenMP:
 *
 * real    0m5.384s
 * user    0m3.656s
 * sys     0m1.756s
 * 
 * ==21740== NVPROF is profiling process 21740, command: ./mmGPU
 * ==21740== Some kernel(s) will be replayed on device 0 in order to collect all events/metrics.
 * ==21740== Replaying kernel "mm$_omp_fn$0" (done)           
 * ==21740== Profiling application: ./mmGPU
 * ==21740== Profiling result:
 * ==21740== Event result:
 * Invocations                                Event Name         Min         Max         Avg       Total
 * Device "GeForce GT 1030 (0)"
 *     Kernel: mm$_omp_fn$0
 *               1                            warps_launched          72          72          72          72
 *
 *               ==21740== Metric result:
 *               Invocations                               Metric Name                        Metric Description         Min         Max         Avg
 *               Device "GeForce GT 1030 (0)"
 *                   Kernel: mm$_omp_fn$0
 *                             1                 warp_execution_efficiency                 Warp Execution Efficiency      86.81%      86.81%      86.81%
 *
 *
 *
 *
 * Tempo, warps_launched e warp_execution_efficiency CUDA: 
 *
 * real    0m0.797s
 * user    0m0.020s
 * sys     0m0.685s
 *
 * ==32252== NVPROF is profiling process 32252, command: ./mmCU
 * ==32252== Some kernel(s) will be replayed on device 0 in order to collect all events/metrics.
 * ==32252== Replaying kernel "mm_cuda(double*, double*, double*, int)" (done)           
 * ==32252== Profiling application: ./mmCU
 * ==32252== Profiling result:
 * ==32252== Event result:
 * Invocations                                Event Name         Min         Max         Avg       Total
 * Device "GeForce GT 1030 (0)"
 *    Kernel: mm_cuda(double*, double*, double*, int)
 * 	          1                            warps_launched         512         512         512         512
 * 
 * 			  ==32252== Metric result:
 * 			  Invocations                               Metric Name                        Metric Description         Min         Max         Avg
 * 			  Device "GeForce GT 1030 (0)"
 * 			      Kernel: mm_cuda(double*, double*, double*, int)
 * 				            1                 warp_execution_efficiency                 Warp Execution Efficiency      78.38%      78.38%      78.38%
 * 
 *
 *
 */



__global__ void mm_cuda(double* a, double* b, double* c, int width) 
{
//  #pragma omp parallel for schedule(dynamic)
//	#pragma omp target map(to:a[0:width*width], b[0:width*width]) map(from:c[0:width*width])
//	#pragma omp teams distribute parallel for simd
	int j = blockIdx.y*blockDim.y+threadIdx.y;
	int i = blockIdx.x*blockDim.x+threadIdx.x;

	if((i < width) && (j < width))
	{	
		double sum = 0;
	    for (int k = 0; k < width; k++) 
		{
			double x = a[i * width + k];
			double y = b[k * width + j];
			sum += x * y;
			//printf("%lf", sum);
		}
		c[i * width + j] = sum;
	}
}

int main()
{
	int width = 100;
	double *a = (double*) malloc (width * width * sizeof(double));
    double *b = (double*) malloc (width * width * sizeof(double));
    double *c = (double*) malloc (width * width * sizeof(double));
  
	//#pragma omp parallel for schedule(dynamic)
    for(int i = 0; i < width; i++) {	  
      for(int j = 0; j < width; j++) {
        a[i*width+j] = i;
        b[i*width+j] = j;
        c[i*width+j] = 0;
      }
    }

	int size = width*width*sizeof(double);
	double *d_a, *d_b, *d_c;

	hipMalloc((void **) &d_a, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	hipMalloc((void **) &d_b, size);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  	hipMalloc((void **) &d_c, size);

	int block_size = 32;
    dim3 dimGrid((width-1)/block_size+1, (width-1)/block_size+1, 1);
    dim3 dimBlock(block_size, block_size, 1);

    mm_cuda<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, width);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
   /*	
    for(int i = 0; i < width; i++) {
      for(int j = 0; j < width; j++) {
        printf("\n c[%d][%d] = %lf",i,j,c[i*width+j]);
      }
    }
	*/

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}
